
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void conv(float *image, float *ker, float *out, int im_size, int k_size, int n_ker){
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int b_id = blockIdx.x;
    
        for(int j = 0; j<k_size; j++){
            for(int k = 0; k<k_size; k++){
                int ker_co = b_id*k_size*k_size + k_size*j + k;
                int im_co = (j+tid_y)*im_size + k + tid_x;
                out[b_id*(im_size - k_size + 1)*(im_size - k_size + 1) + tid_y*(im_size - k_size + 1) + tid_x] += image[im_co] * ker[ker_co];
            }
        }
//     out[b_id*28*28 + tid_y*28 + tid_x] = tanh(out[b_id*28*28 + tid_y*28 + tid_x]);
}

// __global__ void conv(float *image, float *ker, float *out, int k_size, int n_ker){
//     int tid_x = threadIdx.x;
//     int tid_y = threadIdx.y;
//     int b_id = blockIdx.x;
    
//         for(int j = 0; j<5; j++){
//             for(int k = 0; k<5; k++){
//                 int ker_co = b_id*25 + 5*j + k;
//                 int im_co = (j+tid_y)*32 + k + tid_x;
//                 out[b_id*28*28 + tid_y*28 + tid_x] += image[im_co] * ker[ker_co];
//             }
//         }
// //     out[b_id*28*28 + tid_y*28 + tid_x] = tanh(out[b_id*28*28 + tid_y*28 + tid_x]);
// }

__global__ void averagePool(float *in, int len_x, int len_y, int len_z, float *out){
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int b_id = blockIdx.x;
    int surf = len_x*len_y;
    
    out[surf/4*b_id + tid_y*len_x/2 + tid_x] = (in[surf*b_id + tid_y*len_x*2 + tid_x*2] + 
                                              in[surf*b_id + tid_y*len_x*2 + tid_x*2 + 1] + 
                                              in[surf*b_id + tid_y*len_x*2 + len_x + tid_x*2] + 
                                              in[surf*b_id + tid_y*len_x*2 + len_x + tid_x*2 + 1])/4;
}

__device__ float activation_tanh(float M){
    return tanh(M);
}

void init(float *image, int L, int n_ker = 1){
    for(int l = 0; l < n_ker; l++){
        for(int i = 0; i<L; i++){
            for(int j = 0; j<L; j++){
                image[l*L*L + i*L + j] = i;//i + j + 2*l;
            }
        }
    }
}

// identity filter
void init_ker(float *ker, int L){
    ker[(L/2)*L + L/2] = 1;
}

// edge detector horizontal
void init_ker2(float *ker, int L){
    ker[(L*L) + (L/2)*L + L/2 - 1] = -1;
    ker[(L*L) + (L/2)*L + L/2 + 1] = 1;
}

// edge detector vertical
void init_ker3(float *ker, int L){
    ker[2*(L*L) + (L/2 - 1)*L + L/2 ] = -1;
    ker[2*(L*L) + (L/2 + 1)*L + L/2] = 1;
}

void print_vec(float *vec, int L){
    for(int i = 0; i<L; i++){
        printf("%f\n", vec[i]);
    }
}

void print_mat(float *vec, int L, int H, int n_layer = 1){
    for(int l = 0; l<n_layer; l++){
        for(int i = 0; i<H; i++){
            for(int j = 0; j<L; j++)
                printf("%2.2f ", vec[l*L*H + i*L + j]);
            printf("\n");
        }
        printf("\n");
    }
}

void testConv(){
    int L = 32;
    int k_size = 5;
    int n_ker = 3;
    int L_out =  (L - k_size + 1) * (L - k_size + 1) * n_ker;
    int L_ker = k_size * k_size * n_ker;
    
    float *image = (float *) malloc(sizeof(float) * L * L);
    float *ker = (float *) malloc(sizeof(float) * L_ker);
    float *out = (float *) malloc(sizeof(float) * L_out);
    
    float *image_d, *ker_d, *out_d;

    init_ker(ker, k_size);
    init_ker2(ker, k_size);
    init_ker3(ker, k_size);
    
    init(image, L);
    
    hipMalloc(&image_d, sizeof(float) * L * L);
    hipMalloc(&ker_d, sizeof(float) * L_ker);
    hipMalloc(&out_d, sizeof(float) * L_out);
    
    hipMemcpy(image_d, image, sizeof(float) * L * L, hipMemcpyHostToDevice);
    hipMemcpy(ker_d, ker, sizeof(float) * L_ker, hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, sizeof(float) * L_out, hipMemcpyHostToDevice);
    
    dim3 threadPerBlock(28, 28);
    
    conv<<<n_ker, threadPerBlock>>>(image_d, ker_d, out_d, L, k_size, n_ker);
    hipMemcpy(out, out_d, sizeof(float) * L_out, hipMemcpyDeviceToHost);
    print_mat(out, 28, n_ker*28);
    hipDeviceSynchronize();
}

void testMaxPool(){
    int L = 32;
    int n_layer = 2;
    int L_out =  L/2;
    int im_size = L * L * n_layer;
    int out_size = L_out * L_out * n_layer;
    
    float *image = (float *) malloc(sizeof(float) * im_size);
    float *out = (float *) malloc(sizeof(float) * out_size);
    
    float *image_d, *out_d;
    
    init(image, L, n_layer);
    
    hipMalloc(&image_d, sizeof(float) * im_size);
    hipMalloc(&out_d, sizeof(float) * out_size);
    
    hipMemcpy(image_d, image, sizeof(float) * im_size, hipMemcpyHostToDevice);
    hipMemcpy(out_d, out, sizeof(float) * out_size, hipMemcpyHostToDevice);
    
    dim3 threadPerBlock(L_out, L_out);
    
    averagePool<<<n_layer, threadPerBlock>>>(image_d, L, L, n_layer, out_d);
    hipMemcpy(out, out_d, sizeof(float) * out_size, hipMemcpyDeviceToHost);
    print_mat(image, L, L, n_layer);
    
    print_mat(out, L_out, L_out, n_layer);
    
    hipDeviceSynchronize();
}
int main(){
//     testMaxPool();
    testConv();
    
    return 0;
}

